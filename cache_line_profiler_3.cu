#include <stdio.h>
#include <hip/hip_runtime.h>

// GPU Kernel
__global__ void P_chasing2(volatile int *A, long long int iterations, long long int *d_tvalue, int stride) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    long long int start_time, end_time;

    // Use a volatile pointer to prevent compiler optimizations
    volatile int *ptr = &A[index];

    for (long long int i = 0; i < iterations; ++i) {
        start_time = clock64();
        index = ptr[index]; // Pointer chasing to generate memory accesses
        ptr[index] = index; // Write to prevent any compiler optimizations
        end_time = clock64();

        // Calculate and write the time taken for each access
        d_tvalue[i] = (end_time - start_time);
    }
}

// Host function to initialize A with a specific stride
void init_cpu_data(int* A, int size, int stride) {
    for (int i = 0; i < size; i++) {
        // Every 'stride' elements are a multiple of the cache line size apart
        A[i] = ((i + stride) % size) * stride;
    }
}

int main() {
    const long long int iterations = 1024;
    const int size = 4096;
    int *h_A = (int *)malloc(size * sizeof(int));
    long long int *h_tvalue = (long long int *)malloc(iterations * sizeof(long long int));

    volatile int *d_A;
    long long int *d_tvalue;
    hipMalloc((void **)&d_A, size * sizeof(int));
    hipMalloc((void **)&d_tvalue, iterations * sizeof(long long int));

    // Set the cache preference and output appropriate message
    hipFuncCache_t cacheConfigs[2] = {hipFuncCachePreferL1, hipFuncCachePreferShared};
    const char *configNames[2] = {"L1 Enabled", "L1 Disabled"};

    for (int config = 0; config < 2; ++config) {
        hipDeviceSetCacheConfig(cacheConfigs[config]);
        printf("CacheConfig,%s\n", configNames[config]);  // Indicate cache configuration

        for (int stride = 1; stride <= 128; stride++) {  // More granular stride values
            init_cpu_data(h_A, size, stride);
            hipMemcpy((void *)d_A, h_A, size * sizeof(int), hipMemcpyHostToDevice);

            P_chasing2<<<1, 1>>>((int *)d_A, iterations, d_tvalue, stride);

            hipMemcpy(h_tvalue, d_tvalue, iterations * sizeof(long long int), hipMemcpyDeviceToHost);

            for (int i = 0; i < iterations; ++i) {
                printf("Stride,%d,Iteration,%d,Time,%lld\n", stride, i, h_tvalue[i]);  // Detailed output
            }
        }
    }

    // Cleanup
    hipFree((void *)d_A);
    hipFree(d_tvalue);
    free(h_A);
    free(h_tvalue);

    return 0;
}
